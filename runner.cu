#include "1-naive.cuh"

#include <iostream>

#define CEIL_DIV(M, N) (M + N - 1) / N

void run_kernel(int kernel_id) {
  // Allocate memory the size of which we are concerned with
  const int PROBLEM_SIZE = 4096;
  const int PROBLEM_MATRIX_SIZE = PROBLEM_SIZE * PROBLEM_SIZE;
  const int PROBLEM_MATRIX_SIZE_BYTES = PROBLEM_MATRIX_SIZE * sizeof(float);

  float* h_M = new float[PROBLEM_MATRIX_SIZE_BYTES];
  float* h_C = new float[PROBLEM_MATRIX_SIZE_BYTES];

  // Initialize h_M with some values;
  for (int i = 0; i < PROBLEM_SIZE; ++i) {
    for (int j = 0; j < PROBLEM_SIZE; ++j) {
      h_M[i * PROBLEM_SIZE + j] = static_cast<float>((i + j) % 5);
    }
  }

  // Malloc on CUDA some arrays then copy over the values
  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc(&d_A, PROBLEM_MATRIX_SIZE_BYTES);
  hipMalloc(&d_B, PROBLEM_MATRIX_SIZE_BYTES);
  hipMalloc(&d_C, PROBLEM_MATRIX_SIZE_BYTES);

  hipMemcpy(d_A, h_M, PROBLEM_MATRIX_SIZE_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_M, PROBLEM_MATRIX_SIZE_BYTES, hipMemcpyHostToDevice);

  // Now we can run our kernels
  dim3 blockDim(16, 16);
  dim3 gridDim(CEIL_DIV(PROBLEM_SIZE, 16), CEIL_DIV(PROBLEM_SIZE, 16));

  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);

  switch (kernel_id) {
    case 0:
      gemm_naive<<<gridDim, blockDim>>>(PROBLEM_SIZE, PROBLEM_SIZE, PROBLEM_SIZE, d_A, d_B, d_C);
      break;
    default:
      throw std::runtime_error("Unexpected kernel_id");
  }
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);

  // Now copy the result back to the host
  hipMemcpy(h_C, d_C, PROBLEM_MATRIX_SIZE_BYTES, hipMemcpyDeviceToHost);

  // Print out the first few entries of the matrix
  std::cout << ">>>Ran kernel_id=" << kernel_id << std::endl;
  std::cout << "h_C[0]=" << h_C[0] << std::endl;
  std::cout << "h_C[1]=" << h_C[1] << std::endl;
  std::cout << "h_C[2]=" << h_C[2] << std::endl;

  float elapsed = 0.f;
  hipEventElapsedTime(&elapsed, start_event, stop_event);
  std::cout << "Kernel ran for " << elapsed << " ms";

  // Free all the memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  delete[] h_M;
  delete[] h_C;
}

int main() {
  run_kernel(0);
} 